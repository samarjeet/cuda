#include<iostream>
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>

int getDeviceProperties(){
  hipDeviceProp_t iProp;
  int nDevices;
  
  hipGetDeviceCount(&nDevices);
  std::cout << "Number of CUDA devices :  " << nDevices << "\n";
  std::cout << "Properties of device : " << nDevices - 1 << "\n";
  hipGetDeviceProperties(&iProp, nDevices-1);
  std::cout << "Device name : " << iProp.name << "\n";
  std::cout << "Number of multiprocessors : " << iProp.multiProcessorCount << "\n";
  std::cout << iProp.totalConstMem/1024.0 << " KB \n";
  std::cout << iProp.sharedMemPerBlock/1024.0 << " KB \n";
  std::cout << iProp.regsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor/32 << "  \n";
  return 0;
}

__global__ void sumMatrixOnGPU2D(float *a, float *b, float *c, int nx, int ny){
  unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int idx = iy * nx + ix;
  if (ix < nx && iy < ny) {
    c[idx] = a[ix] + b[ix];
  }
}

int main(int argc, char *argv[]){
  int nx = 1<<14;
  int ny = 1<<14;


  float *d_a, *d_b, *d_c;
  float *h_a, *h_b, *h_c;

  h_a = (float*)malloc(sizeof(float)*nx*ny);
  h_b = (float*)malloc(sizeof(float)*nx*ny);
  h_c = (float*)malloc(sizeof(float)*nx*ny);

  for (int i=0 ; i < nx; ++i){
      for (int j=0; j < ny; ++j){
          h_a[j * nx + i ] = 1.0;
          h_b[j * nx + i ] = 2.0;
      }
  }

  hipMalloc(&d_a, sizeof(float)*nx*ny);
  hipMalloc(&d_b, sizeof(float)*nx*ny);
  hipMalloc(&d_c, sizeof(float)*nx*ny);

  hipMemcpy(&d_a, &h_a, sizeof(float)*nx*ny, hipMemcpyHostToDevice);
  hipMemcpy(&d_b, &h_b, sizeof(float)*nx*ny, hipMemcpyHostToDevice);


  int dimx, dimy;
  if (argc > 2) {
    dimx = atoi(argv[1]);
    dimy = atoi(argv[2]);
  }
  else {
    dimx = 32;
    dimy = 32;
  }

  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1 )/ block.x, (ny + block.y -1)/block.y);

  //size_t iStart, eElaps;
  hipDeviceSynchronize();
  auto iStart = std::chrono::system_clock::now();
  sumMatrixOnGPU2D<<<grid, block >>>(d_a, d_b, d_c, nx, ny);
  auto iEnd = std::chrono::system_clock::now();
  std::chrono::duration<double> diff = iEnd - iStart;
  std::cout << "Time : " << diff.count() << "s \n";
  hipMemcpy(&h_c, &d_c, sizeof(float)*nx*ny, hipMemcpyDeviceToHost);
  return 0;
}
