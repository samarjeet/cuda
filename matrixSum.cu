#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include <hip/hip_runtime.h>

int getDeviceProperties(){
  hipDeviceProp_t iProp;
  int nDevices;
  
  hipGetDeviceCount(&nDevices);
  std::cout << "Number of CUDA devices :  " << nDevices << "\n";
  std::cout << "Properties of device : " << nDevices - 1 << "\n";
  hipGetDeviceProperties(&iProp, nDevices-1);
  std::cout << "Device name : " << iProp.name << "\n";
  std::cout << "Number of multiprocessors : " << iProp.multiProcessorCount << "\n";
  std::cout << iProp.totalConstMem/1024.0 << " KB \n";
  std::cout << iProp.sharedMemPerBlock/1024.0 << " KB \n";
  std::cout << iProp.regsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor/32 << "  \n";
  return 0;
}

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int nx, int ny){
  unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

  unsigned int idx = iy * nx + ix;
  if (ix < nx && iy < ny) {
    c[ix] = a[ix] + b[ix];
  }
}

int main(int argc, char *argv[]){
  int nx = 1<<14;
  int ny = 1<<14;

  int dimx, dimy;
  if (argc > 2) {
    dimx = atoi(argv[1]);
    dimy = atoi(argv[2]);
  }
  else {
    dimx = 32;
    dimy = 32;
  }

  dim3 block(dimx, dimy);
  dim3 grid((nx + block.x - 1 )/ block.x, (ny + block.y -1)/block.y);

  return 0; 
}
