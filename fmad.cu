#include<iostream>
#include<hip/hip_runtime.h>
#include<stdio.h>

__global__
void kernel_fmad(float* ptr, float* res){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  //if (tid == 0)
    *res = *ptr * *ptr + *ptr;
  printf("Result is : %f \n", *res);
}

__global__
void kernel_fmad2(float ptr1, float ptr2, float* res){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  //if (tid == 0)
    *res = ptr1 * ptr1 + ptr2;
}

void fmad(float* ptr, float* res){
  *res = *ptr * *ptr + *ptr;
}

int main(int argc, char** argv){
  float *host_f, *host_res;
  float *device_f, *device_res ;
  float host_device_res;

  host_f = (float*)malloc(sizeof(float));
  host_res = (float*)malloc(sizeof(float));
  *host_f = 12.6f;
  //cudaMalloc((void **)&device_f, sizeof(float));
  hipMalloc((void **)&device_res, sizeof(float));
  
  //cudaMemcpy(device_f, host_f, sizeof(float), cudaMemcpyHostToDevice);

  fmad(host_f, host_res);
  //kernel_fmad<<<1,32>>>(device_f, device_res);
  float x = 12.6f;
  float y = 12.6f;
  kernel_fmad2<<<1,32>>>(x,y, device_res);
  hipMemcpy(&host_device_res, device_res, sizeof(float), hipMemcpyDeviceToHost);
  printf("Host kernel : %f \n", *host_res);
  printf("Device kernel : %f \n", host_device_res);
  //cudaFree(device_f);
  free(host_f);
  return 0;
}
