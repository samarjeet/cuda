#include<iostream>
#include<hip/hip_runtime.h>

/*

  Using CUDA warp level primitives
  all threads in a warp are executed in SIMT fashioni

  compute bound workloads

  criteria
  ------------
  performance 
  numerical accuracy and
  thread-safety


  3 topics 
  ------------
  floating point operations
  intrinsic and standard functions
  atomic operations

  
*/

__global__
void float_precision(float* f, double* d){
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  *f = 12.1;
  *d = 12.1;
}

void precision(){
  std::cout << "Testing single and double precision differences in CUDA. \n";

  // Accuracy
  double d = 12.1;
  float f  = 12.1;

  printf("Host single precision representation of 12. 1 : %.32f\n", f);
  printf("Host double precision representation of 12. 1 : %.32f\n", d);

  double* device_d;
  float*  device_f;
  double host_device_d;
  float host_device_f;

  hipMalloc((void **) &device_f, sizeof(float));
  hipMalloc((void **) &device_d, sizeof(double));
  float_precision<<<1,32>>>(device_f, device_d);

  hipMemcpy(&host_device_f, device_f, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&host_device_d, device_d, sizeof(double), hipMemcpyDeviceToHost);

  printf("Device single precision representation of 12. 1 : %.32f\n", host_device_f);
  printf("Device double precision representation of 12. 1 : %.32f\n", host_device_d);

  if (host_device_d == d) printf("Double representation is same for host and device.\n");
  if (host_device_f == f) printf("Float representation is same for host and device.\n");

  printf("Difference between single and double precision : %.32f\n", d-f);
  // Performance


  // Correctness
}
int main(int argc, char** argv){
  precision();
  return 0;
}
