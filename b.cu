#include<iostream>
#include <hip/hip_runtime.h>

int main(){
  std::cout << "Hello World from CUDA\n";
  hipDeviceProp_t iProp;
  int nDevices;
  
  hipGetDeviceCount(&nDevices);
  std::cout << "Number of CUDA devices :  " << nDevices << "\n";
  std::cout << "Properties of device : " << nDevices - 1 << "\n";
  hipGetDeviceProperties(&iProp, nDevices-1);
  std::cout << "Device name : " << iProp.name << "\n";
  std::cout << "Number of multiprocessors : " << iProp.multiProcessorCount << "\n";
  std::cout << iProp.totalConstMem/1024.0 << " KB \n";
  std::cout << iProp.sharedMemPerBlock/1024.0 << " KB \n";
  std::cout << iProp.regsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor/32 << "  \n";
  std::cout << iProp.major << " " << iProp.minor << "  \n";
  return 0;
}
