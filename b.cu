#include<iostream>
#include <hip/hip_runtime.h>

int main(){
  std::cout << "Hello World from CUDA\n";
  hipDeviceProp_t iProp;
  hipGetDeviceProperties(&iProp, 2);
  std::cout << iProp.multiProcessorCount << "\n";
  std::cout << iProp.totalConstMem/1024.0 << " KB \n";
  std::cout << iProp.sharedMemPerBlock/1024.0 << " KB \n";
  std::cout << iProp.regsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerBlock << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor << "  \n";
  std::cout << iProp.maxThreadsPerMultiProcessor/32 << "  \n";
  return 0;
}
